#include "hip/hip_runtime.h"
//
// Created by thomas on 25/09/23.
//

#include <cstdio>
#include "hello.h"

__global__ void sayHello() {
    printf("Hello world from the GPU!\n");
}

int helloTest() {
    printf("Hello world from the CPU!\n");

    sayHello<<<1,1>>>();
    hipDeviceSynchronize();

    return 0;
}
